#include "hip/hip_runtime.h"

#include"z.h"

#defineBLOCK_DIM_X64

#defineN(x)(erf((x)/sqrt(2.0f))/2+0.5f)

__global__ void gpuBlackScholes(float* call,float* S,float* X,float* T,float* r,float* sigma,int len){

  int ii=threadIdx.x+blockDim.x*blockIdx.x;
  if(ii>len){
    return;
  }
  float d1=
  (log(S[ii]/X[ii])+(r[ii]+(sigma[ii]*sigma[ii])/2)*T[ii])/(sigma[ii]*sqrt(T[ii]));
  float d2=d1-sigma[ii]*sqrt(T[ii]);

  call[ii]=S[ii]*N(d1)-X[ii]*exp(-r[ii]*T[ii])*N(d2);
}

void BlackSholes(zMemory_t out,zMemory_t S,zMemory_t X,zMemory_t T,zMemory_t r,zMemory_t sigma){
  size_t len=zMemory_getFlattenedLength(S);
  dim3 blockDim(BLOCK_DIM_X);
  dim3 gridDim(zCeil(len,blockDim.x));
  zState_t st=zMemory_getState(out);
  hipStream_t strm=zState_getComputeStream(st,zMemory_getId(out));
  gpuBlackScholes<<<gridDim,blockDim,0,strm>>>
    ((float*)zMemory_getDeviceMemory(out),
    (float*)zMemory_getDeviceMemory(S),
    (float*)zMemory_getDeviceMemory(X),
    (float*)zMemory_getDeviceMemory(T),
    (float*)zMemory_getDeviceMemory(r),
    (float*)zMemory_getDeviceMemory(sigma),
    len);
  return;
}

int main(int argc,char* argv[]){
  size_t dim=atoi(argv[1]);
  zMemoryGroup_t S=zReadFloatArray(st,"S",1,&dim);
  zMemoryGroup_t X=zReadFloatArray(st,"X",1,&dim);
  zMemoryGroup_t T=zReadFloatArray(st,"T",1,&dim);
  zMemoryGroup_t r=zReadFloatArray(st,"r",1,&dim);
  zMemoryGroup_t q=zReadFloatArray(st,"q",1,&dim);
  zMemoryGroup_t out=zMemoryGroup_new(st,zMemoryType_float,1,&dim);
  zMapGroupFunction_t mapFun=zMapGroupFunction_new(st,"blackScholes",BlackSholes);
  zMap(st,mapFun,out,S,X,T,r,q);
  zWriteFloatArray(st,"out",out);
  return 0;
}